
#include <hip/hip_runtime.h>
#include <chrono>
#include <unordered_map>
#include <string>
#include <algorithm>
#include <iostream>
#include <vector>
#include <utility>
#include "stdio.h"
#include "stdlib.h"


class MyTimer {
public:
    void start(const std::string& name) {
    	hipDeviceSynchronize();
        time_points[name] = std::chrono::high_resolution_clock::now();
    }

    void stop(const std::string& name) {
    	hipDeviceSynchronize();
        auto end_time = std::chrono::high_resolution_clock::now();
        durations[name] += std::chrono::duration<double, std::milli>(end_time - time_points[name]).count();
        count[name] += 1;
    }

    double elapsedMilliseconds(const std::string& name, const std::string& mode = "average") const {
        if (mode == "average")
            return durations.at(name)/count.at(name);
        else 
            return durations.at(name);
    }

    void printAllTimes(
        int iteration,
        int world_size,
        int local_rank,
        const char* log_folder = nullptr
    ) const {
        char* prefix = new char[100];
		sprintf(prefix, "\nit=%d,ws:%d,rk=%d  -->\n", iteration, world_size, local_rank);
		char* filename = new char[100];
		sprintf(filename, "%s/time_ws=%d_rk=%d.log", log_folder, world_size, local_rank);
        // merge the above two lines into one line
        // printf("printAllTimes %d %d %d %s, prefix: %s, filename: %s", iteration, world_size, local_rank, log_folder, prefix, filename);


        std::vector<std::pair<std::string, double>> sortedTimes(durations.begin(), durations.end());
        std::sort(sortedTimes.begin(), sortedTimes.end(), 
            [](const auto& a, const auto& b) {
                return a.first < b.first;
            }
        );
        std::cout << prefix << std::endl;
        for (const auto& pair : sortedTimes) {
            std::cout << pair.first << " time: " << elapsedMilliseconds(pair.first, "sum") << " ms" << std::endl;
        }
        //save in file
        FILE *fp;
        fp = fopen(filename, "a");
        fprintf(fp, "%s", prefix);
        for (const auto& pair : sortedTimes) {
            fprintf(fp, "%s time: %f ms\n", pair.first.c_str(), elapsedMilliseconds(pair.first, "sum"));
        }
        //clean up
        fclose(fp);
        delete[] prefix;
        delete[] filename;
    }

protected:
    std::unordered_map<std::string, std::chrono::time_point<std::chrono::high_resolution_clock>> time_points;
    std::unordered_map<std::string, double> durations;
    std::unordered_map<std::string, int> count;
};

class MyTimerOnGPU {
public:
    void start(const std::string& name) {
        hipEventCreate(&start_events[name]);
        hipEventRecord(start_events[name]);
        durations[name] = -1;
    }

    void stop(const std::string& name) {
        hipEventCreate(&stop_events[name]);
        hipEventRecord(stop_events[name]);
    }

    double elapsedMilliseconds(const std::string& name, const std::string& mode = "average") {
        if (durations[name] > -0.5) {
            return durations[name];
        }
        hipEventSynchronize(stop_events.at(name));
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start_events.at(name), stop_events.at(name));
        durations[name] = (double)milliseconds;
        return (double)milliseconds;
    }

    void printAllTimes(
        int iteration,
        int world_size,
        int local_rank,
        const char* log_folder = nullptr
    ) {
        char* prefix = new char[100];
		sprintf(prefix, "\nit=%d,ws:%d,rk=%d  -->\n", iteration, world_size, local_rank);
		char* filename = new char[100];
		sprintf(filename, "%s/gpu_time_ws=%d_rk=%d.log", log_folder, world_size, local_rank);
        // merge the above two lines into one line

        std::vector<std::pair<std::string, double>> sortedTimes(durations.begin(), durations.end());
        std::sort(sortedTimes.begin(), sortedTimes.end(), 
            [](const auto& a, const auto& b) {
                return a.first < b.first;
            }
        );
        std::cout << prefix << std::endl;
        for (const auto& pair : sortedTimes) {
            std::cout << pair.first << " time: " << elapsedMilliseconds(pair.first, "sum") << " ms" << std::endl;
        }
        //save in file
        FILE *fp;
        fp = fopen(filename, "a");
        fprintf(fp, "%s", prefix);
        for (const auto& pair : sortedTimes) {
            fprintf(fp, "%s time: %f ms\n", pair.first.c_str(), elapsedMilliseconds(pair.first, "sum"));
        }
        //clean up
        fclose(fp);
        delete[] prefix;
        delete[] filename;
    }

protected:
    std::unordered_map<std::string, double> durations;
    std::unordered_map<std::string, hipEvent_t> start_events;
    std::unordered_map<std::string, hipEvent_t> stop_events;
};